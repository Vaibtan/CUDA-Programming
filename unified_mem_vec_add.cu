#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <>

#define EXIT_SIG 0

__global__ void vector_add_um(int *__restrict a, int *__restrict b, int *__restrict c, int LEN) {
    // GLOBAL THREAD ID
    int thrd_id = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (thrd_id < LEN) c[thrd_id] = a[thrd_id] + b[thrd_id];
}

inline void constexpr vector_init(int *a, int *b, int LEN) {
    for (int __iter = 0; __iter < LEN; ++__iter) {
        a[__iter] = rand() % 100;
        b[__iter] = rand() % 32;
    }
}

inline void constexpr error_check(int *a, int *b, int *c, int LEN) {
    for (int __iter = 0; __iter < LEN; ++__iter) assert(c[__iter] == a[__iter] + b[__iter]);
}

int main() {
    //device ID for prefetching calls
    int id = hipGetDevice(&id);
    const int static N = 1 << 16;
    size_t bytes = sizeof(int) * N;
    int *a, *b, *c;

    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    hipMemAdvise(a, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(b, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemPrefetchAsync(c, bytes, id);

    vector_init(a, b, N);

    // data is mostly going to be read from and only occasionally written to
    hipMemAdvise(a, bytes, hipMemAdviseSetReadMostly, id);
    hipMemAdvise(b, bytes, hipMemAdviseSetReadMostly, id);
    hipMemPrefetchAsync(a, bytes, id);
    hipMemPrefetchAsync(b, bytes, id);

    const int static BLOCK_SZ = 256;
    const int static GRID_SZ = (int)ceil(N / BLOCK_SZ);

    // LAUNCH KERNEL ON DEFAULT STREAM W/O SHARED_MEM
    vector_add_um<<<GRID_SZ, BLOCK_SZ>>>(a, b, c, N);
    hipDeviceSynchronize();

    //PREFETCH c TO HOST
    hipMemPrefetchAsync(a, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(b, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);
    error_check(a, b, c, N);

    hipFree(a);
    hipFree(b);
    hipFree(c);

    printf("COMPLETED SUCCESSFULLY\n");
    return EXIT_SIG;
}